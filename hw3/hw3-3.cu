
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

constexpr int ceil(int a, int b) { return (a + b - 1) / b; }
static const int INF = ((1 << 30) - 1);
static const int BLK_WIDTH = 2;

struct HostData
{
    int nV, nE;
    int nPadV; // padding elements to fit the width of the block
    int *H_Dist = NULL;

    HostData(int nVertex, int nEdge) : nV(nVertex), nE(nEdge)
    {
        nPadV = ceil(nV, BLK_WIDTH) * BLK_WIDTH;
        hipHostMalloc(&H_Dist, nPadV * nPadV * sizeof(int), hipHostMallocDefault);
    }
    ~HostData()
    {
        hipFree(H_Dist);
    }
};

/* util */
inline int h_index(int i, int j, int row_size) { return i * row_size + j; }
__device__ inline int d_index(int i, int j, int row_size) { return i * row_size + j; }

/* debug */
void h_printMatrix(int *arr, int width);
__device__ void d_printMatrix(int *arr, int width);

/* IO */
HostData *input(char *inFileName);
void output(char *outFileName, const HostData *hData);

/* APSP */
void blk_FW(int nV, int nPadV, int *H_Dist);
__global__ void naiveCudaFWKernal(int *D_Dist, int k, int nPadV);
__global__ void naiveCudaFW(int *D_Dist, int nV, int nPadV);
__global__ void calPhase1(int *D_Dist, int pivot, int nPadV);
__global__ void calPhase2(int *D_Dist, int pivot, int nPadV, int devID);
__global__ void calPhase3(int *D_Dist, int pivot, int nPadV, int devID);

int main(int argc, char *argv[])
{
    /* input */
    HostData *hData = input(argv[1]);
    printf("%d\n", hData->nPadV);

    /* blocked Floyd-Washall */
    // h_printMatrix(H_Dist, nPadV); // debug
    blk_FW(hData->nV, hData->nPadV, hData->H_Dist);

    /* naive Floyd-Washall */
    // dim3 dimGrid(nPadV / BLK_WIDTH, nPadV / BLK_WIDTH);
    // dim3 dimBlock(BLK_WIDTH, BLK_WIDTH);
    // cudaFuncSetCacheConfig(naiveCudaFWKernal, cudaFuncCachePreferL1);
    // for (int k = 0; k < hData->nV; ++k)
    //     naiveCudaFWKernal <<< dimGrid, dimBlock >>> (D_Dist, k, nPadV);

    /* output */
    // h_printMatrix(H_Dist, nPadV); // debug
    output(argv[2], hData);
    delete hData;

    return 0;
}

/* blocked FW */
void blk_FW(int nV, int nPadV, int *H_Dist)
{
    const int nBlk = ceil(nV, BLK_WIDTH);
    dim3 dimGridPhase1(1, 1);
    dim3 dimGridPhase2(nPadV / BLK_WIDTH, 2); // blockIdx.y is the flag that marks it as a column or a row.
    dim3 dimGridPhase3(nPadV / BLK_WIDTH, nPadV / BLK_WIDTH);
    dim3 dimBlk(BLK_WIDTH, BLK_WIDTH);

    int *D0_Dist = NULL;
    int *D1_Dist = NULL;
    hipSetDevice(0);
    hipHostGetDevicePointer(&D0_Dist, H_Dist, 0);
    hipSetDevice(1);
    hipHostGetDevicePointer(&D1_Dist, H_Dist, 0);

    // h_printMatrix(H_Dist, nPadV); // for debug
    for (int pivot = 0; pivot < nBlk; ++pivot)
    {
        /* phase 1 */
        calPhase1<<<dimGridPhase1, dimBlk>>>(D0_Dist, pivot, nPadV);
        hipDeviceSynchronize();

        /* phase 2 */
        hipSetDevice(0);
        calPhase2<<<dimGridPhase2, dimBlk>>>(D0_Dist, pivot, nPadV, 0);
        hipSetDevice(1);
        calPhase2<<<dimGridPhase2, dimBlk>>>(D1_Dist, pivot, nPadV, 1);
        hipSetDevice(0);
        hipDeviceSynchronize();
        hipSetDevice(1);
        hipDeviceSynchronize();

        /* phase 3 */
        hipSetDevice(0);
        calPhase3<<<dimGridPhase3, dimBlk>>>(D0_Dist, pivot, nPadV, 0);
        hipSetDevice(1);
        calPhase3<<<dimGridPhase3, dimBlk>>>(D1_Dist, pivot, nPadV, 1);
        hipSetDevice(0);
        hipDeviceSynchronize();
        hipSetDevice(1);
        hipDeviceSynchronize();
    }
    // h_printMatrix(H_Dist, nPadV); // for debug
}
__global__ void calPhase1(int *D_Dist, int pivot, int nPadV)
{
    const int idx = threadIdx.x;            // share memory index
    const int idy = threadIdx.y;            // share memory index
    const int v1 = pivot * BLK_WIDTH + idy; // global memory index
    const int v2 = pivot * BLK_WIDTH + idx; // global memory index

    /* load */
    const int s_index = d_index(idy, idx, BLK_WIDTH);
    // const int g_index = d_index(v1, v2, nPadV);
    __shared__ int S_Dist_Blk_Update[BLK_WIDTH * BLK_WIDTH];
    S_Dist_Blk_Update[s_index] = D_Dist[d_index(v1, v2, nPadV)];
    __syncthreads();

/* debug, check data in share memory */
// if (idx == 0 && idy == 0)
// {
//     printf("blkIdx.x=%d, blkIdx.y=%d\n", blockIdx.x, blockIdx.y);
//     for (int i = 0; i < BLK_WIDTH; ++i)
//         for (int j = 0; j < BLK_WIDTH; ++j)
//             printf("S_Dist_Blk_Update[%d][%d]=%d\n", idx + i, idy + j, S_Dist_Blk_Update[d_index(idx + i, idy + j, BLK_WIDTH)]);
// }

/* computing */
#pragma unroll
    for (int k = 0; k < BLK_WIDTH; ++k)
    {
        const int new_dist = S_Dist_Blk_Update[d_index(idy, k, BLK_WIDTH)] + S_Dist_Blk_Update[d_index(k, idx, BLK_WIDTH)];
        if (new_dist < S_Dist_Blk_Update[s_index])
            S_Dist_Blk_Update[s_index] = new_dist;
        __syncthreads();
    }

    /* store */
    D_Dist[d_index(v1, v2, nPadV)] = S_Dist_Blk_Update[s_index];
}
__global__ void calPhase2(int *D_Dist, int pivot, int nPadV, int devID)
{
    /* exception */
    if ((blockIdx.x == pivot) || // phase 1
        (blockIdx.y != devID)    // 2-devices
    )
        return;

    /* variables */
    const int idx = threadIdx.x;            // share memory index
    const int idy = threadIdx.y;            // share memory index
    const int v1 = pivot * BLK_WIDTH + idy; // global memory index
    const int v2 = pivot * BLK_WIDTH + idx; // global memory index

    /* load */
    const int s_index = d_index(idy, idx, BLK_WIDTH);
    const int g_index = (blockIdx.y == 0) ? d_index(v1, blockIdx.x * BLK_WIDTH + idx, nPadV) : d_index(blockIdx.x * BLK_WIDTH + idy, v2, nPadV);
    __shared__ int S_Dist_Blk_Base[BLK_WIDTH * BLK_WIDTH];
    __shared__ int S_Dist_Blk_Update[BLK_WIDTH * BLK_WIDTH];
    S_Dist_Blk_Base[s_index] = D_Dist[d_index(v1, v2, nPadV)];
    S_Dist_Blk_Update[s_index] = D_Dist[g_index];
    __syncthreads();

    /* debug, check data in share memory */
    // if (idx == 0 && idy == 0 && blockIdx.x == 1 && blockIdx.y == 0)
    // {
    //     for (int i = 0; i < BLK_WIDTH; ++i)
    //         for (int j = 0; j < BLK_WIDTH; ++j)
    //             printf("blkIdx.x=%d, blkIdx.y=%d, S_Dist_Blk_Base[%d][%d]=%d\n", blockIdx.x, blockIdx.y, idx + i, idy + j, S_Dist_Blk_Base[d_index(idx + i, idy + j, BLK_WIDTH)]);
    //     for (int i = 0; i < BLK_WIDTH; ++i)
    //         for (int j = 0; j < BLK_WIDTH; ++j)
    //             printf("blkIdx.x=%d, blkIdx.y=%d, S_Dist_Blk_Update[%d][%d]=%d\n", blockIdx.x, blockIdx.y, idx + i, idy + j, S_Dist_Blk_Update[d_index(idx + i, idy + j, BLK_WIDTH)]);
    // }

    /* computing */
    if (blockIdx.y == 0)
    {
#pragma unroll
        for (int k = 0; k < BLK_WIDTH; ++k)
        {
            const int new_dist = S_Dist_Blk_Base[d_index(idy, k, BLK_WIDTH)] + S_Dist_Blk_Update[d_index(k, idx, BLK_WIDTH)];
            if (new_dist < S_Dist_Blk_Update[s_index])
                S_Dist_Blk_Update[s_index] = new_dist;
            __syncthreads();
        }
    }
    else
    {
#pragma unroll
        for (int k = 0; k < BLK_WIDTH; ++k)
        {
            // const int new_dist = S_Dist_Blk_Update[d_index(idy, k, BLK_WIDTH)] + S_Dist_Blk_Base[d_index(k, idx, BLK_WIDTH)];
            if (S_Dist_Blk_Update[d_index(idy, k, BLK_WIDTH)] + S_Dist_Blk_Base[d_index(k, idx, BLK_WIDTH)] < S_Dist_Blk_Update[s_index])
                S_Dist_Blk_Update[s_index] = S_Dist_Blk_Update[d_index(idy, k, BLK_WIDTH)] + S_Dist_Blk_Base[d_index(k, idx, BLK_WIDTH)];
            __syncthreads();
        }
    }

    /* store */
    D_Dist[g_index] = S_Dist_Blk_Update[s_index];
}
__global__ void calPhase3(int *D_Dist, int pivot, int nPadV, int devID)
{
    /* exception */
    if ((blockIdx.x == pivot || blockIdx.y == pivot) ||                        // phase 1, 2
        (devID == 0 && blockIdx.x > pivot || devID == 1 && blockIdx.x < pivot) // 2 devices
    )
        return;

    /* variables */
    const int idx = threadIdx.x;                 // share memory index
    const int idy = threadIdx.y;                 // share memory index
    const int v1 = blockIdx.y * BLK_WIDTH + idy; // global memory index
    const int v2 = blockIdx.x * BLK_WIDTH + idx; // global memory index
    const int pv1 = pivot * BLK_WIDTH + idy;     // global memory index
    const int pv2 = pivot * BLK_WIDTH + idx;     // global memory index

    /* load */
    const int s_index = d_index(idy, idx, BLK_WIDTH);
    // const int g_index = d_index(v1, v2, nPadV);
    int min_dist = D_Dist[d_index(v1, v2, nPadV)];
    __shared__ int S_Dist_Blk_Row[BLK_WIDTH * BLK_WIDTH];
    __shared__ int S_Dist_Blk_Col[BLK_WIDTH * BLK_WIDTH];
    S_Dist_Blk_Row[s_index] = D_Dist[d_index(pv1, v2, nPadV)];
    S_Dist_Blk_Col[s_index] = D_Dist[d_index(v1, pv2, nPadV)];
    __syncthreads();

/* debug, check data in share memory */
// if (idx == 0 && idy == 0 && blockIdx.x == 1 && blockIdx.y == 2)
// {
//     for (int i = 0; i < BLK_WIDTH; ++i)
//         for (int j = 0; j < BLK_WIDTH; ++j)
//             printf("blkIdx.x=%d, blkIdx.y=%d, S_Dist_Blk_Row[%d][%d]=%d\n", blockIdx.x, blockIdx.y, idx + i, idy + j, S_Dist_Blk_Row[d_index(idx + i, idy + j, BLK_WIDTH)]);
//     for (int i = 0; i < BLK_WIDTH; ++i)
//         for (int j = 0; j < BLK_WIDTH; ++j)
//             printf("blkIdx.x=%d, blkIdx.y=%d, S_Dist_Blk_Col[%d][%d]=%d\n", blockIdx.x, blockIdx.y, idx + i, idy + j, S_Dist_Blk_Col[d_index(idx + i, idy + j, BLK_WIDTH)]);
// }

/* computing */
#pragma unroll
    for (int k = 0; k < BLK_WIDTH; ++k)
    {
        const int new_dist = S_Dist_Blk_Col[d_index(idy, k, BLK_WIDTH)] + S_Dist_Blk_Row[d_index(k, idx, BLK_WIDTH)];
        if (new_dist < min_dist)
            min_dist = new_dist;
        // __syncthreads(); // no data dependency here
    }

    /* store */
    D_Dist[d_index(v1, v2, nPadV)] = min_dist;
}

/* naive FW */
__global__ void naiveCudaFWKernal(int *D_Dist, int k, int nPadV)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int new_dist = D_Dist[d_index(i, k, nPadV)] + D_Dist[d_index(k, j, nPadV)];

    if (new_dist < D_Dist[i * nPadV + j])
        D_Dist[i * nPadV + j] = new_dist;
}
__global__ void naiveCudaFW(int *D_Dist, int nV, int nPadV) // result is not correct
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;

    for (int k = 0; k < nV; ++k)
    {
        const int new_dist = D_Dist[d_index(i, k, nPadV)] + D_Dist[d_index(k, j, nPadV)];
        if (new_dist < D_Dist[i * nPadV + j])
            D_Dist[i * nPadV + j] = new_dist;
        __syncthreads();
    }
}

/* IO */
HostData *input(char *infile)
{
    FILE *file = fopen(infile, "rb");
    if (!file)
    {
        printf("Could not open %s\n", infile);
        fflush(stdout);
        exit(EXIT_FAILURE);
    }

    int nV, nE;
    size_t vf = fread(&nV, sizeof(int), 1, file);
    size_t ef = fread(&nE, sizeof(int), 1, file);
    HostData *hData(NULL);
    hData = new HostData(nV, nE);

    for (int i = 0; i < hData->nPadV; ++i)
    {
        for (int j = 0; j < hData->nPadV; ++j)
        {
            if (i == j && i < nV && j < nV)
            {
                (hData->H_Dist)[h_index(i, j, hData->nPadV)] = 0;
            }
            else
            {
                (hData->H_Dist)[h_index(i, j, hData->nPadV)] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < nE; ++i)
    {
        size_t pf = fread(pair, sizeof(int), 3, file);
        (hData->H_Dist)[h_index(pair[0], pair[1], hData->nPadV)] = pair[2];
    }
    fclose(file);

    return hData;
}
void output(char *outFileName, const HostData *hData)
{
    FILE *outfile = fopen(outFileName, "w");
    for (int i = 0; i < hData->nV; ++i)
    {
        for (int j = 0; j < hData->nV; ++j)
        {
            if ((hData->H_Dist)[h_index(i, j, hData->nPadV)] >= INF)
                (hData->H_Dist)[h_index(i, j, hData->nPadV)] = INF;
        }
        fwrite(&(hData->H_Dist)[h_index(i, 0, hData->nPadV)], sizeof(int), hData->nV, outfile);
    }
    fclose(outfile);
}

/* debug */
void h_printMatrix(int *arr, int width)
{
    printf("------------------------------------------------------------------------------\n");
    for (int x = 0; x < width; ++x)
    {
        for (int y = 0; y < width; ++y)
            printf("%12d\t", arr[x * width + y]);
        printf("\n");
    }
    printf("------------------------------------------------------------------------------\n");
}
__device__ void d_printMatrix(int *arr, int width)
{
    printf("------------------------------------------------------------------------------\n");
    for (int x = 0; x < width; ++x)
    {
        for (int y = 0; y < width; ++y)
            printf("%12d\t", arr[x * width + y]);
        printf("\n");
    }
    printf("------------------------------------------------------------------------------\n");
}